/**********************************************************************/
// An unoptimized Naive N-Body solver for Gravity Simulations         //
// G is assumed to be 1.0                                             //
// Course Material for HPCSE-II, Spring 2019, ETH Zurich              //
// Authors: Sergio Martin                                             //
// License: Use if you like, but give us credit.                      //
/**********************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "string.h"
#include <chrono>

void checkCUDAError(const char *msg);

//Eex9Koh4pha-
__global__ void forceKernel(double* xPos, double* yPos, double* zPos, double* mass, double* xFor, double* yFor, double* zFor, size_t N)
{
 size_t m = blockIdx.x*blockDim.x+threadIdx.x;

 for (size_t i = 0; i < N; i++) if (i != m)
 {
  double xDist = xPos[m] - xPos[i];
  double yDist = yPos[m] - yPos[i];
  double zDist = zPos[m] - zPos[i];
  double r     = sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
  xFor[m] += xDist*mass[m]*mass[i] / (r*r*r);
  yFor[m] += yDist*mass[m]*mass[i] / (r*r*r);
  zFor[m] += zDist*mass[m]*mass[i] / (r*r*r);
 }
}

int main(int argc, char* argv[])
{
 size_t N0 = 80;
 size_t N  = N0*N0*N0;

 // Initializing N-Body Problem

 double* xPos   = (double*) calloc (N, sizeof(double));
 double* yPos   = (double*) calloc (N, sizeof(double));
 double* zPos   = (double*) calloc (N, sizeof(double));
 double* xFor   = (double*) calloc (N, sizeof(double));
 double* yFor   = (double*) calloc (N, sizeof(double));
 double* zFor   = (double*) calloc (N, sizeof(double));
 double* mass   = (double*) calloc (N, sizeof(double));

 size_t current = 0;
 for (size_t i = 0; i < N0; i++)
 for (size_t j = 0; j < N0; j++)
 for (size_t k = 0; k < N0; k++)
 {
  xPos[current] = i;
  yPos[current] = j;
  zPos[current] = k;
  mass[current] = 1.0;
  xFor[current] = 0.0;
  yFor[current] = 0.0;
  zFor[current] = 0.0;
  current++;
 }

 // Allocating and initializing GPU memory

 double* dxPos; hipMalloc((void **) &dxPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dyPos; hipMalloc((void **) &dyPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dzPos; hipMalloc((void **) &dzPos,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dxFor; hipMalloc((void **) &dxFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dyFor; hipMalloc((void **) &dyFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dzFor; hipMalloc((void **) &dzFor,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");
 double* dmass; hipMalloc((void **) &dmass,  sizeof(double) * N); checkCUDAError("Unable to allocate storage on the device");

 hipMemcpy(dxPos, xPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dyPos, yPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dzPos, zPos, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dxFor, xFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dyFor, yFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dzFor, zFor, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");
 hipMemcpy(dmass, mass, sizeof(double) * N, hipMemcpyHostToDevice); checkCUDAError("Failed Initial Conditions Memcpy");

 // Calculating Kernel Geometry
 size_t threadsPerBlock  = 1024;
 size_t blocksPerGrid    = ceil(double (((double)N) / ((double)threadsPerBlock)));

 // Running Force-calculation kernel
 auto startTime = std::chrono::system_clock::now();
 forceKernel<<<blocksPerGrid, threadsPerBlock>>>(dxPos, dyPos, dzPos, dmass, dxFor, dyFor, dzFor, N); checkCUDAError("Failed Force Kernel");
 hipDeviceSynchronize();
 auto endTime = std::chrono::system_clock::now();

 hipMemcpy(xFor, dxFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");
 hipMemcpy(yFor, dyFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");
 hipMemcpy(zFor, dzFor, sizeof(double) * N, hipMemcpyDeviceToHost); checkCUDAError("Failed Final Conditions Memcpy");

 double netForce = 0.0;
 double absForce = 0.0;
 for (size_t i = 0; i < N; i++) netForce += xFor[i] + yFor[i] + zFor[i];
 for (size_t i = 0; i < N; i++) absForce += abs(xFor[i] + yFor[i] + zFor[i]);

 printf("     Net Force: %.6f\n", netForce);
 printf("Absolute Force: %.6f\n", absForce);

 if (isfinite(netForce) == false)      { printf("Verification Failed: Net force is not a finite value!\n"); exit(-1); }
 if (fabs(netForce) > 0.00001)         { printf("Verification Failed: Force equilibrium not conserved!\n"); exit(-1); }
 if (isfinite(absForce) == false)      { printf("Verification Failed: Absolute Force is not a finite value!\n"); exit(-1); }

 printf("Time: %.8fs\n", std::chrono::duration<double>(endTime-startTime).count());
 return 0;
}

void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err)
 {
  fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err) );
  exit(EXIT_FAILURE);
 }
}
