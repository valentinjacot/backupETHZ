#include "hip/hip_runtime.h"
/**********************************************************************/
// A now optimized Multigrid Solver for the Heat Equation             //
// Course Material for HPCSE-II, Spring 2019, ETH Zurich              //
// Authors: Sergio Martin, Georgios Arampatzis                        //
// License: Use if you like, but give us credit.                      //
/**********************************************************************/

#include <stdio.h>
#include <math.h>
#include <limits>
#include "heat2d_gpu.hpp"
#include "string.h"
#include <chrono>

pointsInfo __p;

void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess !=err )
 {
   fprintf(stderr," CUDA Error: %s: %s. \n",msg, hipGetErrorString(err));
	exit(-1);	
 }
}
//__global__ void dgemm(double *A, double *B, double *C, size_t N)

int main(int argc, char* argv[])
{
 double tolerance = 1e-0; // L2 Difference Tolerance before reaching convergence.
 size_t N0 = 10; // 2^N0 + 1 elements per side

 // Multigrid parameters -- Find the best configuration!
 size_t gridCount       = N0-1;     // Number of Multigrid levels to use
 size_t downRelaxations = 5; // Number of Relaxations before restriction
 size_t upRelaxations   = 0;   // Number of Relaxations after prolongation
 
  
 gridLevel* g = generateInitialConditions(N0, gridCount);
	
  gridLevel* d_g;
  hipMalloc(&d_g, sizeof(gridLevel) * gridCount);
  hipMemcpy(d_g,&g,sizeof(gridLevel) * gridCount, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  
  
 auto startTime = std::chrono::system_clock::now();
 while (g[0].L2NormDiff > tolerance)  // Multigrid solver start
 {
  
  
  applyJacobi(d_g, 0, downRelaxations); // Relaxing the finest grid first
  calculateResidual(g, 0); // Calculating Initial Residual

  for (size_t grid = 1; grid < gridCount; grid++) // Going down the V-Cycle
  {
   applyRestriction(g, grid); // Restricting the residual to the coarser grid's solution vector (f)
   applyJacobi(g, grid, downRelaxations); // Smoothing coarser level
   calculateResidual(g, grid); // Calculating Coarse Grid Residual
  }

  for (size_t grid = gridCount-1; grid > 0; grid--) // Going up the V-Cycle
  {
   applyProlongation(g, grid); // Prolonging solution for coarser level up to finer level
   applyJacobi(g, grid, upRelaxations); // Smoothing finer level
  }

  calculateL2Norm(g, 0); // Calculating Residual L2 Norm
 }  // Multigrid solver end

  hipDeviceSynchronize();
  hipMemcpy(g,&d_g,sizeof(gridLevel) * gridCount, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
//  __syncthreads(); //  hipDeviceSynchronize();


 auto endTime = std::chrono::system_clock::now();
 totalTime = std::chrono::duration<double>(endTime-startTime).count();
 printTimings(gridCount);
 printf("L2Norm: %.4f\n",  g[0].L2Norm);
 freeGrids(g, gridCount);
 return 0;
}

//kernel function. Works only on the assigned thread Eex9Koh4pha-
__global__ void applyJacobiKernel(gridLevel* g, size_t l, double h1, double h2)
{
 size_t myRow = blockIdx.y*blockDim.y + threadIdx.y;
 size_t myCol = blockIdx.x*blockDim.x + threadIdx.x;
 if(myRow >= g[l].N-1 || myCol >= g[l].N-1)return; 
 if(myRow < 1 || myCol < 1 )return;
  //for (size_t i = 1; i < g[l].N-1; i++)
   //for (size_t j = 1; j < g[l].N-1; j++) // Perform a Jacobi Iteration
 g[l].U[myRow][myCol] = (g[l].Un[myRow-1][myCol] + g[l].Un[myRow+1][myCol] + g[l].Un[myRow][myCol-1] + g[l].Un[myRow][myCol+1] + g[l].f[myRow][myCol]*h2)*h1;
__syncthreads();
}

void applyJacobi(gridLevel* d_g, size_t l, size_t relaxations)
{
 int N = d_g[l].N;

 dim3 threadsPerBlock(32,32);
 dim3 blocksPerGrid(N/32,N/32);//depends on the size of the problem. int or dim3
 auto t0 = std::chrono::system_clock::now();
 double h1 = 0.25;
 double h2 = d_g[l].h*d_g[l].h;
 for (size_t r = 0; r < relaxations; r++)
 {
  double** tmp = d_g[l].Un; d_g[l].Un = d_g[l].U; d_g[l].U = tmp;
  applyJacobiKernel<<< blocksPerGrid,threadsPerBlock >>>(d_g, l, h1, h2);
 }
auto t1 = std::chrono::system_clock::now();
 smoothingTime[l] += std::chrono::duration<double>(t1-t0).count();
}

__global__ void calculateResidualKernel(gridLevel* g, size_t l, double h2)
{
 size_t myRow = blockIdx.y*blockDim.y + threadIdx.y;
 size_t myCol = blockIdx.x*blockDim.x + threadIdx.x;
 if(myRow >= g[l].N-1 || myCol >= g[l].N-1)return; 
 if(myRow < 1 || myCol < 1 )return;
 g[l].Res[myRow][myCol] = g[l].f[myRow][myCol] + (g[l].U[myRow-1][myCol] + g[l].U[myRow+1][myCol] - 4*g[l].U[myRow][myCol] + g[l].U[myRow][myCol-1] + g[l].U[myRow][myCol+1]) * h2;
}

void calculateResidual(gridLevel* g, size_t l)
{
 int N = g[l].N;
 dim3 threadsPerBlock(32,32);
 dim3 blocksPerGrid(N/32,N/32);//depends on the size of the problem. int or dim3
 auto t0 = std::chrono::system_clock::now();
 double h2 = 1.0 / pow(g[l].h,2);
 calculateResidualKernel<<< blocksPerGrid,threadsPerBlock >>>(g, l, h2);
 auto t1 = std::chrono::system_clock::now();
 residualTime[l] += std::chrono::duration<double>(t1-t0).count();
}

__global__ void calculateL2NormKernel(gridLevel* g, size_t l)
{
 double tmp = 0.0;
 size_t i = blockIdx.y*blockDim.y + threadIdx.y;
 size_t j = blockIdx.x*blockDim.x + threadIdx.x;
 if(i >= g[l].N-1 || j >= g[l].N-1)return; 
 if(i < 1 || j < 1 )return;
   g[l].Res[i][j] = g[l].Res[i][j]*g[l].Res[i][j];
   __syncthreads();
   tmp += g[l].Res[i][j];
   __syncthreads();
}

void calculateL2Norm(gridLevel* g, size_t l)
{
 int N = d_g[l].N;
 dim3 threadsPerBlock(32,32);
 dim3 blocksPerGrid(N/32,N/32);//depends on the size of the problem. int or dim3
 double tmp = 0.0;
 auto t0 = std::chrono::system_clock::now();

 calculateL2NormKernel<<< blocksPerGrid,threadsPerBlock >>>(d_g, l);

 auto t1 = std::chrono::system_clock::now();
 g[l].L2Norm = sqrt(tmp);
 g[l].L2NormDiff = fabs(g[l].L2NormPrev - g[l].L2Norm);
 g[l].L2NormPrev = g[l].L2Norm;
 
 L2NormTime[l] += std::chrono::duration<double>(t1-t0).count();
}

__global__ void applyRestrictionKernel(gridLevel* g, size_t l)
{
 double tmp = 0.0;
 size_t i = blockIdx.y*blockDim.y + threadIdx.y;
 size_t j = blockIdx.x*blockDim.x + threadIdx.x;
 if(i >= g[l].N-1 || j >= g[l].N-1)return; 
 if(i < 1 || j < 1 )return;
 g[l].f[i][j] = ( 1.0*( g[l-1].Res[2*i-1][2*j-1] + g[l-1].Res[2*i-1][2*j+1] + g[l-1].Res[2*i+1][2*j-1]   + g[l-1].Res[2*i+1][2*j+1] )   +
             2.0*( g[l-1].Res[2*i-1][2*j]   + g[l-1].Res[2*i][2*j-1]   + g[l-1].Res[2*i+1][2*j]     + g[l-1].Res[2*i][2*j+1] ) +
             4.0*( g[l-1].Res[2*i][2*j] ) ) * 0.0625;
 __syncthreads();
  g[l].U[i][j] = 0;// Resetting U vector for the coarser level before smoothing -- Find out if this is really necessary.
}

void applyRestriction(gridLevel* g, size_t l)
{ 
 int N = g[l].N;
 dim3 threadsPerBlock(32,32);
 dim3 blocksPerGrid(N/32,N/32);//depends on the size of the problem. int or dim3
 auto t0 = std::chrono::system_clock::now();

 applyRestrictionKernel<<< blocksPerGrid,threadsPerBlock >>>(d_g, l);

 auto t1 = std::chrono::system_clock::now();
 restrictionTime[l] += std::chrono::duration<double>(t1-t0).count();
}

__global__ void applyProlongationKernel(gridLevel* g, size_t l)
{
 double tmp = 0.0;
 size_t i = blockIdx.y*blockDim.y + threadIdx.y;
 size_t j = blockIdx.x*blockDim.x + threadIdx.x;
 if(i < 1 || j < 1 );
 if(myRow < g[l].N-1 || myCol >= g[l].N-1)
}

void applyProlongation(gridLevel* g, size_t l)
{
 auto t0 = std::chrono::system_clock::now();

 for (size_t i = 1; i < g[l].N-1; i++)
  for (size_t j = 1; j < g[l].N-1; j++)
    g[l-1].U[2*i][2*j] += g[l].U[i][j];

 for (size_t i = 1; i < g[l].N; i++)
  for (size_t j = 1; j < g[l].N-1; j++)
   g[l-1].U[2*i-1][2*j] += ( g[l].U[i-1][j] + g[l].U[i][j] ) *0.5;

 for (size_t i = 1; i < g[l].N-1; i++)
  for (size_t j = 1; j < g[l].N; j++)
   g[l-1].U[2*i][2*j-1] += ( g[l].U[i][j-1] + g[l].U[i][j] ) *0.5;

 for (size_t i = 1; i < g[l].N; i++)
  for (size_t j = 1; j < g[l].N; j++)
   g[l-1].U[2*i-1][2*j-1] += ( g[l].U[i-1][j-1] + g[l].U[i-1][j] + g[l].U[i][j-1] + g[l].U[i][j] ) *0.25;

 auto t1 = std::chrono::system_clock::now();
 prolongTime[l] += std::chrono::duration<double>(t1-t0).count();
}

gridLevel* generateInitialConditions(size_t N0, size_t gridCount)
{
 // Default values:
 __p.nCandles = 4;
 std::vector<double> pars;
 pars.push_back(0.228162);
 pars.push_back(0.226769);
 pars.push_back(0.437278);
 pars.push_back(0.0492324);
 pars.push_back(0.65915);
 pars.push_back(0.499616);
 pars.push_back(0.59006);
 pars.push_back(0.0566329);
 pars.push_back(0.0186672);
 pars.push_back(0.894063);
 pars.push_back(0.424229);
 pars.push_back(0.047725);
 pars.push_back(0.256743);
 pars.push_back(0.754483);
 pars.push_back(0.490461);
 pars.push_back(0.0485152);

//	std::vector<double>* d_pars;
//	hipMalloc(&d_pars, pars.size()*sizeof(double));
//	hipMemcpy(d_pars,&pars,pars.size()*sizeof(double), hipMemcpyDeviceToHost);

 // Allocating Timers
 smoothingTime = (double*) calloc (gridCount, sizeof(double));
 residualTime = (double*) calloc (gridCount, sizeof(double));
 restrictionTime = (double*) calloc (gridCount, sizeof(double));
 prolongTime = (double*) calloc (gridCount, sizeof(double));
 L2NormTime = (double*) calloc (gridCount, sizeof(double));

 // Allocating Grids
 gridLevel* g = (gridLevel*) malloc(sizeof(gridLevel) * gridCount);
 for (size_t i = 0; i < gridCount; i++)
 {
  g[i].N = pow(2, N0-i) + 1;
  g[i].h = 1.0/(g[i].N-1);

  g[i].U   = (double**) malloc(sizeof(double*) * g[i].N); for (size_t j = 0; j < g[i].N ; j++) g[i].U[j]   = (double*) malloc(sizeof(double) * g[i].N);
  g[i].Un  = (double**) malloc(sizeof(double*) * g[i].N); for (size_t j = 0; j < g[i].N ; j++) g[i].Un[j]  = (double*) malloc(sizeof(double) * g[i].N);
  g[i].Res = (double**) malloc(sizeof(double*) * g[i].N); for (size_t j = 0; j < g[i].N ; j++) g[i].Res[j] = (double*) malloc(sizeof(double) * g[i].N);
  g[i].f   = (double**) malloc(sizeof(double*) * g[i].N); for (size_t j = 0; j < g[i].N ; j++) g[i].f[j]   = (double*) malloc(sizeof(double) * g[i].N);

  g[i].L2Norm = 0.0;
  g[i].L2NormPrev = std::numeric_limits<double>::max();
  g[i].L2NormDiff = std::numeric_limits<double>::max();
 }

 // Initial Guess
 for (size_t i = 0; i < g[0].N; i++) for (size_t j = 0; j < g[0].N; j++) g[0].U[i][j] = 1.0;

 // Boundary Conditions
 for (size_t i = 0; i < g[0].N; i++) g[0].U[0][i]        = 0.0;
 for (size_t i = 0; i < g[0].N; i++) g[0].U[g[0].N-1][i] = 0.0;
 for (size_t i = 0; i < g[0].N; i++) g[0].U[i][0]        = 0.0;
 for (size_t i = 0; i < g[0].N; i++) g[0].U[i][g[0].N-1] = 0.0;

 // F
 for (size_t i = 0; i < g[0].N; i++)
 for (size_t j = 0; j < g[0].N; j++)
 {
  double h = 1.0/(g[0].N-1);
  double x = i*h;
  double y = j*h;

  g[0].f[i][j] = 0.0;

  for (size_t c = 0; c < __p.nCandles; c++)
  {
   double c3 = pars[c*4  + 0]; // x0
   double c4 = pars[c*4  + 1]; // y0
   double c1 = pars[c*4  + 2]; c1 *= 100000;// intensity
   double c2 = pars[c*4  + 3]; c2 *= 0.01;// Width
   g[0].f[i][j] += c1*exp(-(pow(c4 - y, 2) + pow(c3 - x, 2)) / c2);
  }
 }
 return g;
}


void freeGrids(gridLevel* g, size_t gridCount)
{
 for (size_t i = 0; i < gridCount; i++)
 {
  for (size_t j = 0; j < g[i].N ; j++) free(g[i].U[j]);
  for (size_t j = 0; j < g[i].N ; j++) free(g[i].f[j]);
  for (size_t j = 0; j < g[i].N ; j++) free(g[i].Res[j]);
  free(g[i].U);
  free(g[i].f);
  free(g[i].Res);
 }
 free(g);
}

void printTimings(size_t gridCount)
{
    double* timePerGrid = (double*) calloc (sizeof(double), gridCount);
    double totalSmoothingTime = 0.0;
    double totalResidualTime = 0.0;
    double totalRestrictionTime = 0.0;
    double totalProlongTime = 0.0;
    double totalL2NormTime = 0.0;

    for (size_t i = 0; i < gridCount; i++) timePerGrid[i] = smoothingTime[i] + residualTime[i] + restrictionTime[i] + prolongTime[i] + L2NormTime[i];
    for (size_t i = 0; i < gridCount; i++) totalSmoothingTime += smoothingTime[i];
    for (size_t i = 0; i < gridCount; i++) totalResidualTime += residualTime[i];
    for (size_t i = 0; i < gridCount; i++) totalRestrictionTime += restrictionTime[i];
    for (size_t i = 0; i < gridCount; i++) totalProlongTime += prolongTime[i];
    for (size_t i = 0; i < gridCount; i++) totalL2NormTime += L2NormTime[i];

    double totalMeasured = totalSmoothingTime + totalResidualTime + totalRestrictionTime + totalProlongTime + totalL2NormTime;

    printf("   Time (s)    "); for (size_t i = 0; i < gridCount; i++) printf("Grid%lu   ", i);                    printf("   Total  \n");
    printf("-------------|-"); for (size_t i = 0; i < gridCount; i++) printf("--------"); printf("|---------\n");
    printf("Smoothing    | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", smoothingTime[i]);    printf("|  %2.3f  \n", totalSmoothingTime);
    printf("Residual     | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", residualTime[i]);     printf("|  %2.3f  \n", totalResidualTime);
    printf("Restriction  | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", restrictionTime[i]);  printf("|  %2.3f  \n", totalRestrictionTime);
    printf("Prolongation | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", prolongTime[i]);      printf("|  %2.3f  \n", totalProlongTime);
    printf("L2Norm       | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", L2NormTime[i]);       printf("|  %2.3f  \n", totalL2NormTime);
    printf("-------------|-"); for (size_t i = 0; i < gridCount; i++) printf("--------"); printf("|---------\n");
    printf("Total        | "); for (size_t i = 0; i < gridCount; i++) printf("%2.3f   ", timePerGrid[i]); printf("|  %2.3f  \n", totalMeasured);
    printf("-------------|-"); for (size_t i = 0; i < gridCount; i++) printf("--------"); printf("|---------\n");
    printf("\n");
    printf("Running Time      : %.3fs\n", totalTime);
}

