#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err)
 {
  fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err) );
  exit(EXIT_FAILURE);
 }
}

#define BLOCKSIZE 1024

__global__ void reduce(unsigned int* dVec, unsigned int* dAux, size_t N)
{
 size_t  tid = threadIdx.x;
 size_t  i = blockIdx.x*blockDim.x + threadIdx.x;
 __shared__ unsigned int i;
 sdata[tid]=dVec[i];
__syncthreads();
 for(size_t s= 1; s<blockDim.x;s*=2){
  if(tid%(s*2) ==0)sdata[tid] +=sdata[tid+s];
  __syncthreads();
 }
 if (tid==0)dAux[blockDim.x] = sdata[0];
}

int main(int argc, char** argv)
{
	unsigned int *vec;
	unsigned int *dVec, *dAux;

  size_t N0 = 32768;
  size_t N = N0*N0;

  vec = (unsigned int*) malloc (sizeof(unsigned int)*N);

  for (size_t i = 0; i < N; i++) vec[i] = i;

  hipMalloc(&dVec,  sizeof(unsigned int)*N); checkCUDAError("Error allocating dVec");
  hipMalloc(&dAux, sizeof(unsigned int)*N); checkCUDAError("Error allocating dAux");
  hipMemcpy(dVec, vec, sizeof(unsigned int)*N, hipMemcpyHostToDevice); checkCUDAError("Error copying vec");
  
  auto startTime = std::chrono::system_clock::now();

  for(size_t n= N; n > 1; n=n/BLOCKSIZE){
	size_t bSize = BLOCKSIZE; if(bSize > n) bSize = n;
	size_t gridSize = ceil((double)n/(double)BLOCKSIZE); if (bSize > n) gridSize =1;
        printf("bSize: %lu -  gridSize: %lu \n", bSize, gridSize);
	reduce<<<gridSize, bSize>>>(dVec, dAux, n); checkCUDAError("Error kernel");
	unsigned int 

  }

  hipDeviceSynchronize();

  auto endTime = std::chrono::system_clock::now();
  
  unsigned int result = 0.0;
  hipMemcpy(&result, dVec, sizeof(unsigned int), hipMemcpyDeviceToHost); checkCUDAError("Error getting result");

  printf("[GPU] Result: %u - Elapsed Time: %fs\n", result, std::chrono::duration<double>(endTime-startTime).count());

  return 0;
}

