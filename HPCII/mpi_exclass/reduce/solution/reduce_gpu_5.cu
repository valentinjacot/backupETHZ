#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

void checkCUDAError(const char *msg)
{
 hipError_t err = hipGetLastError();
 if( hipSuccess != err)
 {
  fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err) );
  exit(EXIT_FAILURE);
 }
}

#define BLOCKSIZE 1024

__device__ void warpReduce(volatile unsigned int* sdata, int tid) {
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
}

__global__ void reduce(unsigned int* dVec, unsigned int* dAux, size_t N)
{
 __shared__ unsigned int sdata[BLOCKSIZE];

 size_t tid = threadIdx.x;
 size_t i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

 sdata[tid] = dVec[i] + dVec[i+blockDim.x];

 __syncthreads();

 for (unsigned int s=blockDim.x/2; s>32; s>>=1)
 {
  if (tid < s) sdata[tid] += sdata[tid + s];
  __syncthreads();
 }

 if (tid < 32) warpReduce(sdata, tid);

 if (tid == 0) dAux[blockIdx.x] = sdata[0];
}

int main(int argc, char** argv)
{
	unsigned int *vec;
	unsigned int *dVec, *dAux;

  size_t N0 = 32768;
  size_t N = N0*N0;

  vec = (unsigned int*) malloc (sizeof(unsigned int)*N);

  for (size_t i = 0; i < N; i++) vec[i] = i;

  hipMalloc(&dVec,  sizeof(unsigned int)*N); checkCUDAError("Error allocating dVec");
  hipMalloc(&dAux, sizeof(unsigned int)*N); checkCUDAError("Error allocating dAux");
  hipMemcpy(dVec, vec, sizeof(unsigned int)*N, hipMemcpyHostToDevice); checkCUDAError("Error copying vec");
  
  auto startTime = std::chrono::system_clock::now();

  for (size_t n = N; n > 1; n = n / BLOCKSIZE)
  {
  	size_t bSize = BLOCKSIZE;
  	size_t gSize = floor((double)n / (2.0*(double)BLOCKSIZE));
  	if (gSize == 0) { gSize = 2; bSize = n/4; }
  	printf("bSize: %lu - gSize: %lu\n", bSize, gSize);
    reduce<<<gSize, bSize>>>(dVec, dAux, n); checkCUDAError("Failed Kernel Launch");
    unsigned int *tmp = dVec; dVec = dAux; dAux = tmp;
  }

  hipDeviceSynchronize();

  auto endTime = std::chrono::system_clock::now();
  
  unsigned int result = 0.0;
  hipMemcpy(&result, dVec, sizeof(unsigned int), hipMemcpyDeviceToHost); checkCUDAError("Error getting result");

  printf("[GPU] Result: %u - Elapsed Time: %fs\n", result, std::chrono::duration<double>(endTime-startTime).count());

  return 0;
}

